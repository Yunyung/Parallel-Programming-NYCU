#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8

__global__ void mandelKernel(
    int *d_out, size_t pitch, 
    float lowerX, float lowerY,
    float stepX, float stepY,
    int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;
    
    int iter;
    for (iter = 0;iter < maxIterations;iter++) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    *((int*)((char*)d_out + thisY*pitch) + thisX) = iter;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *h_out, *d_out; // Result on host and device
    int size = resX * resY * sizeof(int);

    // Allocate memory on host and device
    size_t pitch;
    hipHostAlloc((void **)&h_out, size, hipHostMallocDefault);
    hipMallocPitch((void **)&d_out, &pitch, resX*sizeof(int), resY);
    
    // CUDA function
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(d_out, pitch,
                                                lowerX, lowerY,
                                                stepX, stepY,
                                                resX, maxIterations);
    
    hipMemcpy2D(h_out, resX*sizeof(int), d_out, pitch, 
                 resX*sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_out, size);

    // Free allocated memory
    hipHostFree(h_out);
    hipFree(d_out);
}
