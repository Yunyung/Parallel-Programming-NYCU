#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8

__global__ void mandelKernel(
    int *d_out,
    float lowerX, float lowerY,
    float stepX, float stepY,
    int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = thisY*resX + thisX;

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;
    
    int iter;
    for (iter = 0;iter < maxIterations;iter++) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    d_out[idx] = iter;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *d_out; // Result on host and device
    int size = resX * resY * sizeof(int);

    // Allocate memory on host and device
    hipHostRegister(img, size, hipHostRegisterMapped);
    hipHostGetDevicePointer(&d_out, img, 0);

    // CUDA function
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(d_out,
                                                 lowerX, lowerY,
                                                 stepX, stepY,
                                                 resX, maxIterations);
    
    hipDeviceSynchronize();

    // Free allocated memory
    hipHostUnregister(img);
}
